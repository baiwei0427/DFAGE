#include "hip/hip_runtime.h"
/*
 * Vinh Dang
 * vqd8a@virginia.edu
 *
 * udfa_gpu.cu
 */

#include "common.h"
#include "udfa_gpu.h"

__global__ void udfa_kernel(
				state_t *input_dfa_state_tables,
				symboln *input,
				unsigned int *pkt_size_vec, unsigned int pkt_size,
				unsigned int *match_count, match_type *match_array, unsigned int match_vec_size,
				unsigned int *accum_dfa_state_table_lengths, unsigned int n_subsets){					
	
	unsigned int dfa_id = threadIdx.x + blockIdx.y * blockDim.x;
	match_type tmp_match;
	
	if (dfa_id >= n_subsets)
		return;
	
	unsigned int shr_match_count = 0;
	
	//cur_pkt_size is the input string length of the packet
	unsigned int cur_pkt_size = pkt_size_vec[blockIdx.x];
	
	//jump to the right input string
	input += (pkt_size * blockIdx.x/fetch_bytes); 

	unsigned int accum_dfa_state_table_length = accum_dfa_state_table_lengths[dfa_id];
	
	state_t current_state = 0;

	//loop over payload
	for(unsigned int p=0; p<cur_pkt_size; p+=fetch_bytes, input++){
		symboln Input_ = *input;//fetch 4 bytes from the input string
		for (unsigned int byt = 0; byt < fetch_bytes; byt++) {
			unsigned int Input = Input_ & 0xFF;//extract 1 byte
			Input_  = Input_ >> 8;//Input_ right-shifted by 8 bits
		
			//query the state table on the input symbol for the next state
			current_state = input_dfa_state_tables [current_state * CSIZE + Input + accum_dfa_state_table_length];
			
			if (current_state < 0) {//Added for matching operation: check if the dst state is an accepting state
				current_state = -current_state;
				//match_offset[match_vec_size*blockIdx.x + shr_match_count + dfa_id*match_vec_size*nstreams] = p;
				//match_states[match_vec_size*blockIdx.x + shr_match_count + dfa_id*match_vec_size*nstreams] = current_state;
				tmp_match.off  = p + byt;
				tmp_match.stat = current_state;
				match_array[shr_match_count + match_vec_size*(blockIdx.x + dfa_id*gridDim.x)] = tmp_match;
				
				shr_match_count = shr_match_count + 1;
			}		
		}
	}
	match_count[blockIdx.x + dfa_id*gridDim.x] = shr_match_count;
}
