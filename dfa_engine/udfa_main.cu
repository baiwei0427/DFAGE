/*
 * Vinh Dang
 * vqd8a@virginia.edu
 *
 * udfa_main.cu
 */

#include <iostream>
#include <fstream>
#include <string>

#include <stdio.h>
#include <sys/time.h>
#include <sys/stat.h>

#include "packets.h"
#include "udfa_host.h"

using namespace std;

size_t getFilesize(const char* filename);
void Usage(void);
bool ParseCommandLine(int argc, char *argv[]);

const char *base_name=NULL;

#ifdef DEBUG
const char *timing_filename = NULL;
const char *blksiz_filename = NULL;
#endif

int total_rules=0;
int blksiz_tuning = 0;
int automata_format = 0;

CommonConfigs cfg;

int main(int argc, char* argv[]){

    unsigned int retval;
    std::vector<FiniteAutomaton *> dfa_vec;
	    
	char char_temp;
    char filename[1500], bufftmp[10];

	struct timeval c1, c2, c3, c4, c5;
	long seconds, useconds;
	double t_alloc, t_kernel, t_collect, t_free, t_DFAload, t_in;
	
#ifdef DEBUG
	double t_exec;
	ofstream fp_timing;
	ofstream fp_blksiz;
#endif	
	
	int rulespergroup, *rulestartvec;
	int blockSize;
	
	// Load DFAs from files and stores in arrays of internal data structure
	gettimeofday(&c1, NULL);
	
	retval = ParseCommandLine(argc, argv);
    
	if(!retval)
		return 0;
	
    unsigned int total_bytes = getFilesize(cfg.get_input_file_name());

	cout<< "-----------------User input info--------------------" << endl;
	cout<< "Total number of rules: " << total_rules << endl;
	cout<< "Total input bytes: "   << total_bytes << endl;	
	unsigned int n_subsets   = cfg.get_groups();	
	unsigned int n_packets   = cfg.get_packets();
	unsigned int packet_size = ((total_bytes%n_packets)==0)?(total_bytes/n_packets):(total_bytes/n_packets+1);
	cout<< "Subgraph(s) (or DFA(s)) combined: "   << n_subsets << endl;
	cout<< "Packet(s): "   << n_packets << endl;
    cout<< "Packet size (bytes): " << packet_size << endl;
    if (automata_format ==0)
        cout << "Automata in binary format" << endl;
    else
        cout << "Automata in MNRL format" << endl;
    if (blksiz_tuning ==0)
        cout << "Blocksize tuning is not enabled" << endl;
    else
        cout << "Blocksize tuning is enabled" << endl;
	
	rulestartvec = (int*)malloc (n_subsets * sizeof(int));

    if ((total_rules%n_subsets)==0)
		rulespergroup = total_rules/n_subsets;
	else
		rulespergroup = total_rules/n_subsets + 1;
	//printf("rulespergroup=%d\n",rulespergroup);
	for (unsigned int i=0; i<n_subsets; i++) {
		rulestartvec[i]=i*rulespergroup;
		//printf("rulestartvec[%d]=%d\n",i,rulestartvec[i]);
	}

	cout << endl;
	cout<< "-----------------Loading DFA(s) from file(s)--------------------" << endl;
	cout << "Loading..." << endl;

	for (unsigned int i = 0; i < n_subsets; i++) {		
				
		strcpy (filename,base_name);
		
		strcat (filename,"_");
		snprintf(bufftmp, sizeof(bufftmp),"%d",n_subsets);
		strcat (filename,bufftmp);
		strcat (filename,"/");	
		snprintf(bufftmp, sizeof(bufftmp),"%d",i+1);
		strcat (filename,bufftmp); //cout<< "filename " << i + 1 << ":"<< filename << endl;
		
		FiniteAutomaton *dfa_tmp = NULL;//cout <<"TEST sizeof(*FiniteAutomaton) = " << sizeof(*dfa_tmp)<<endl;

		dfa_tmp = load_dfa_file(filename, i, automata_format);
		
		if(!dfa_tmp){
			printf("Error while loading DFA on the device\n");
			return 0;
		}
		dfa_vec.push_back(dfa_tmp);
	}

	cout << "\nDFA loading done!!!\n\n";
	
	for (unsigned int i = 0; i < n_subsets; i++) {
		if (i!=n_subsets-1) cout << "Sub-ruleset "<< i + 1 << ": Rules: " << rulestartvec[i+1] - rulestartvec[i] <<", States: "<< cfg.get_state_count(i) << endl;	
	    else cout << "Sub-ruleset "<< i + 1 << ": Rules: " << total_rules - rulestartvec[i] <<", States: "<< cfg.get_state_count(i) << endl;	
	}    
	gettimeofday(&c2, NULL);
		
	printf("-----------------Starting dfa execution--------------------\n");
    	
	// open input stream file    
	ifstream fp(cfg.get_input_file_name());
#ifdef DEBUG
	if (timing_filename != NULL)//and timing file
		fp_timing.open(timing_filename,ios::binary | ios::out);
	if (blksiz_filename != NULL)//and timing file
		fp_blksiz.open(blksiz_filename,ios::binary | ios::out);
#endif
		
	unsigned int processed_packets = 0;
{	
	gettimeofday(&c3, NULL);
	
	Packets packets;
    vector<unsigned char> payload;
    vector<unsigned int> payload_count;

	// Read input stream file		
	//cout << "Fixed-size packet processing" << endl;
	unsigned int cnt2=0;
	unsigned int cnt=0;
	if (fp){								
		while ( fp.get(char_temp) ){
			cnt2++;
			payload.push_back(char_temp);
			cnt++;					
			if (cnt==packet_size){				
				//version 2 -- note: padding to each packet if packet_size is not evenly divided by fetch_bytes (e.g. 4, 8)
				if ( (cnt%fetch_bytes) != 0 ) {
					for (unsigned int i = 0; i < (fetch_bytes-(cnt%fetch_bytes)); i++)
						payload.push_back(0);
					packets.set_padded_bytes(fetch_bytes-(cnt%fetch_bytes));
				}				
				packets.add_packet(payload);
				payload_count.push_back(payload.size());//cout << payload.size() << endl;
				processed_packets++;
				payload.clear();
				cnt=0;
			}
		}
		if ((cnt>0)&&(cnt<packet_size)){
			//version 2 -- note: padding to each packet if packet_size is not evenly divided by fetch_bytes (e.g. 4, 8)
			if ( (cnt%fetch_bytes) != 0 ) {
				for (unsigned int i = 0; i < (fetch_bytes-(cnt%fetch_bytes)); i++)
					payload.push_back(0);
				packets.set_padded_bytes(fetch_bytes-(cnt%fetch_bytes));
			}
			packets.add_packet(payload);
			payload_count.push_back(payload.size());//cout << payload.size() << endl;
			processed_packets++;
			payload.clear();
			cnt=0;
		}
	}
	else{
		cout<< "Cannot open input file" << endl;				
	}
	cout << "Number of processed packets: "<< processed_packets << " and total number of bytes: "<< cnt2 << endl;
	for (unsigned int i = 0; i < processed_packets; i++){
		cout << "Packet "<< i << ": " << payload_count[i] << " bytes (padding included)"<< endl;				
	}			
	// End of Fixed-size packet processing
	
	//ofstream myfile2 ("./data/packet_cnts.txt");			
	//for (unsigned int i = 0; i < processed_packets; i++){
	//	myfile2 << payload_count[i] << endl;
	//}
	//myfile2.close();
		
	gettimeofday(&c4, NULL);
		
	//cout << "UDFA!!!" << endl;
	retval = udfa_run(dfa_vec, packets, n_subsets, packet_size, rulestartvec, &t_alloc, &t_kernel, &t_collect, &t_free, &blockSize, blksiz_tuning);	
					
	gettimeofday(&c5, NULL);
}
	cout << "----------------- Kernel execution done -----------------" << endl;

	seconds  = c2.tv_sec  - c1.tv_sec;
	useconds = c2.tv_usec - c1.tv_usec;
    t_DFAload   = ((double)seconds * 1000 + (double)useconds/1000.0);
	
	seconds  = c4.tv_sec  - c3.tv_sec;
	useconds = c4.tv_usec - c3.tv_usec;
    t_in     = ((double)seconds * 1000 + (double)useconds/1000.0);
#ifdef DEBUG	
	seconds  = c5.tv_sec  - c4.tv_sec;
	useconds = c5.tv_usec - c4.tv_usec;
    t_exec   = ((double)seconds * 1000 + (double)useconds/1000.0);
	
    printf("udfa.cu: t_exec= %lf(ms)\n", t_exec);
#endif	

    printf("Execution times: DFA loading (from text): %lf(ms), Input stream loading: %lf(ms), GPU mem alloc: %lf(ms), GPU kernel execution: %lf(ms), Result collecting: %lf(ms), GPU mem release: %lf(ms)\n", t_DFAload, t_in, t_alloc, t_kernel, t_collect, t_free);
	
#ifdef DEBUG	
	//Write timing result to file
	double t_DFAs[7];
	t_DFAs[0] = t_alloc;
	t_DFAs[1] = t_kernel;
	t_DFAs[2] = t_collect;
	t_DFAs[3] = t_free;
	t_DFAs[4] = t_DFAload;
	t_DFAs[5] = t_in;
	t_DFAs[6] = t_exec;
	
	if (timing_filename != NULL)
		fp_timing.write((char *)t_DFAs, 7*sizeof(double));
	if (blksiz_filename != NULL)
		fp_blksiz.write((char *)&blockSize, sizeof(int));
#endif
	
	// close the file	
	fp.close();

#ifdef DEBUG	
	if (timing_filename != NULL)
		fp_timing.close();
	if (blksiz_filename != NULL)
		fp_blksiz.close();
#endif
	
    for (unsigned int i = 0; i < n_subsets; i++) {
		delete dfa_vec[i];
	}
	
	cfg.get_controller().dealloc_host_all();
	
	hipDeviceReset();//Explicitly destroys and cleans up all resources associated with the current device in the current process. Note that this function will reset the device immediately. It is the caller's responsibility to ensure that the device is not being accessed by any other host threads from the process when this function is called.
	//To prevent strange memory leak in some machines (or drivers)
	
	free(rulestartvec);
	
	return 0;
}

bool ParseCommandLine(int argc, char *argv[])
{
	int CurrentItem = 1;
	int retVal;

	while (CurrentItem < argc)
	{

		if (strcmp(argv[CurrentItem], "-a") == 0)
		{
			CurrentItem++;
			base_name=argv[CurrentItem];
			CurrentItem++;
			continue;
		}

		if (strcmp(argv[CurrentItem], "-i") == 0)
		{
			CurrentItem++;
			char *input_filename = NULL;
			input_filename=argv[CurrentItem];
			cfg.set_input_file_name(input_filename);
			CurrentItem++;
			continue;
		}

		if (strcmp(argv[CurrentItem], "-p") == 0)
		{
			CurrentItem++;
			unsigned int parallel_packets;
			retVal = sscanf(argv[CurrentItem],"%d", &parallel_packets);
			cfg.set_packets(parallel_packets);
			if(retVal!=1 || parallel_packets < 1 ){
				printf("Invalid parallel_packets number: %s\n", argv[CurrentItem]);
				return false;
			}
			CurrentItem++;
			continue;
		}
		
		if (strcmp(argv[CurrentItem], "-T") == 0)
		{
			CurrentItem++;
			unsigned int threads_per_block;
			retVal = sscanf(argv[CurrentItem],"%d", &threads_per_block);
			cfg.set_threads_per_block(threads_per_block);
			if(retVal!=1 || threads_per_block < 1 ){
				printf("Invalid THREADS_PER_BLOCK number: %s\n", argv[CurrentItem]);
				return false;
			}
			CurrentItem++;
			continue;
		}
#ifdef DEBUG		
		if (strcmp(argv[CurrentItem], "-f") == 0)
		{
			CurrentItem++;
			timing_filename=argv[CurrentItem];
			CurrentItem++;
			continue;
		}
		
		if (strcmp(argv[CurrentItem], "-ft") == 0)
		{
			CurrentItem++;
			blksiz_filename=argv[CurrentItem];
			CurrentItem++;
			continue;
		}		
#endif
        if (strcmp(argv[CurrentItem], "-g") == 0)
		{
			CurrentItem++;
			unsigned int groups;
			retVal = sscanf(argv[CurrentItem],"%d", &groups);
			cfg.set_groups(groups);
			if(retVal!=1 || groups < 1 ){
				printf("Invalid SUB-RULESETS number: %s\n", argv[CurrentItem]);
				return false;
			}
			CurrentItem++;
			continue;
		}
		
		if (strcmp(argv[CurrentItem], "-N") == 0)
			{
				CurrentItem++;
				retVal = sscanf(argv[CurrentItem],"%d", &total_rules);
				if(retVal!=1 || total_rules < 1 ){
					printf("Invalid TOTAL_RULES number: %s\n", argv[CurrentItem]);
					return false;
				}
				CurrentItem++;
				continue;
		}

		if (strcmp(argv[CurrentItem], "-O") == 0)
			{
				CurrentItem++;
				retVal = sscanf(argv[CurrentItem],"%d", &blksiz_tuning);
				if(retVal!=1 || blksiz_tuning > 1 ){
					printf("Invalid blksiz_tuning param: %s\n", argv[CurrentItem]);
					return false;
				}
				CurrentItem++;
				continue;
		}

		if (strcmp(argv[CurrentItem], "-m") == 0)
			{
				CurrentItem++;
				retVal = sscanf(argv[CurrentItem],"%d", &automata_format);
				if(retVal!=1 || automata_format > 1 ){
					printf("Invalid automata_format param: %s\n", argv[CurrentItem]);
					return false;
				}
				CurrentItem++;
				continue;
		}
		
		/*if (strcmp(argv[CurrentItem], "-P") == 0)
		{
			CurrentItem++;
			print_parameters = 1;
			continue;
		}*/

		if (strcmp(argv[CurrentItem], "-h") == 0 ||
				strcmp(argv[CurrentItem], "-?") == 0)
		{
			CurrentItem++;
			Usage();
			return false;
		}
	}


	return true;
}

void Usage(void) {
    char string[]= "USAGE: ./dfa_engine [OPTIONS] \n" \
					 "\t-a <file> :   automata name (must NOT contain the file extension)\n" \
					 "\t-i <file> :   input file (with file extension)\n"  \
					 "\t-T <n>    :   number of threads per block (overwritten if block size tuning feature is used)\n" \
					 "\t-g <n>    :   number of graphs (or DFAs) to be executed (default: 1)\n" \
					 "\t-p <n>    :   number of parallel packets to be examined (default: 1)\n"\
					 "\t-N <n>    :   total number of rules (subgraphs)\n" \
					 "\t-O <n>    :   0 - block size tuning not enabled; 1 - block size tuned (optional, default: 0 - not tuned)\n" \
					 "\t-m <n>    :   0 - automata in binary format; 1 - automata in MNRL format (optional, default: 0 - binary)\n" \
#ifdef DEBUG
					 "\t-f <name> :   timing result filename (optional, default: empty)\n" \
					 "\t-ft <name>:   blocksize filename (optional, default: empty)\n" \
#endif
					 "\t-h        :   prints this message\n" \
					 "Ex:\t./dfa_engine -a ./data/simple -i ./data/simple.input -T 1 -g 1 -p 1 -N 3\n" \
					 "\t./dfa_engine -a ./data/simple -i ./data/simple.input -T 1 -g 1 -p 1 -N 3 -m 1\n" \
					 "\t./dfa_engine -a ./data/simpletwo -i ./data/simpletwo.input -T 1 -g 2 -p 1 -N 6 -m 1\n" \
					 "\t./dfa_engine -a ./data/simpletwo -i ./data/simpletwo.input -T 2 -g 2 -p 1 -N 6 -m 1 -O 1\n";
    fprintf(stderr, "%s", string);
}

/**
 * Get the size of a file.
 * @return The filesize, or 0 if the file does not exist.
 */
 size_t getFilesize(const char* filename) {
    struct stat st;
    if(stat(filename, &st) != 0) {
        return 0;
    }
    return st.st_size;   
}