#include "hip/hip_runtime.h"
/*
 * Vinh Dang
 * vqd8a@virginia.edu
 *
 * udfa_host.cu
 */

#include <cstdlib>
#include <cassert>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#include <libgen.h>
#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>

#include "packets.h"
#include "common.h"
#include "mem_controller.h"
#include "udfa_host.h"
#include "udfa_gpu.h"
				
using namespace std;

extern CommonConfigs cfg;

/*--------------------------------------------------------------------------------------------------*/
#ifdef TEXTURE_MEM_USE //Texture memory: DFA STATE TABLE
texture<state_t, hipTextureType1D, hipReadModeElementType> tex_dfa_state_tables;
__global__ void udfa_kernel_texture(symboln *input,
									unsigned int *pkt_size_vec, unsigned int pkt_size,
									unsigned int *match_count, match_type *match_array, unsigned int match_vec_size,
									unsigned int *accum_dfa_state_table_lengths, unsigned int n_subsets);
#endif
/*--------------------------------------------------------------------------------------------------*/
void GPUMemInfo() {
   size_t free_byte ;
   size_t total_byte ;
   hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
   if ( hipSuccess != cuda_status ){   
      printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );   
      exit(1);   
   }
   double free_db  = (double)free_byte ;
   double total_db = (double)total_byte ;
   double used_db  = total_db - free_db ;

   printf("GPU memory usage: used = %lf MB, free = %lf MB, total = %f MB\n", used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}
/*--------------------------------------------------------------------------------------------------*/
unsigned int udfa_run(std::vector<FiniteAutomaton *> fa, Packets &packets, unsigned int n_subsets, unsigned int packet_size, int *rulestartvec, double *t_alloc, double *t_kernel, double *t_collect, double *t_free, int *blocksize, int blksiz_tuning){

    struct timeval c0, c1, c2, c3, c4;
    long seconds, useconds;
    unsigned int *h_match_count, *d_match_count;
    match_type   *h_match_array, *d_match_array;
   
    ofstream fp_report;
    char filename[200], bufftmp[10];
   
    state_t *d_dfa_state_tables;
    symbol *d_input;
    unsigned int *d_pkt_size;
    size_t max_shmem=0;
    unsigned int   *accum_dfa_state_table_lengths;//Note: arrays contain accumulated values
    unsigned int *d_accum_dfa_state_table_lengths;
   
	//cout << "------------- Preparing to launch kernel ---------------" << endl;
	//cout << "Packets (Streams or Number of CUDA blocks in x-dimension): " << packets.get_payload_sizes().size() << endl;
	
	//cout << "Accumulated number of symbol per packet (stream): ";
	//for (int i = 0; i < packets.get_payload_sizes().size(); i++)
	//	cout << packets.get_payload_sizes()[i] << " ";
	//cout << endl;
    
	//cout << "Threads per block: " << cfg.get_threads_per_block() << endl;
		
	for (unsigned int i = 0; i < n_subsets; ++i) {
		cout << "Graph (DFA) " << i+1 << endl;
		cout << "   + State count: " << cfg.get_state_count(i) << endl;
		cout << endl;
	}
    
	gettimeofday(&c0, NULL);

	unsigned int tmp_avg_count = packets.get_payload_sizes()[packets.get_payload_sizes().size()-1]/packets.get_payload_sizes().size()*60/n_subsets;//just for now, size of each match array for each packet//????????
	
	cout << "Maximum matches allowed:  " << (tmp_avg_count*packets.get_payload_sizes().size()*n_subsets) << endl;
	
	h_match_array         = (match_type*)malloc ((tmp_avg_count*packets.get_payload_sizes().size()) * n_subsets * sizeof(match_type));//just for now
    h_match_count         = (unsigned int*)malloc ((              packets.get_payload_sizes().size()) * n_subsets * sizeof(unsigned int));//just for now  
    accum_dfa_state_table_lengths = (unsigned int*)malloc (n_subsets * sizeof(unsigned int));
			
	hipMalloc( (void **) &d_match_array,  (tmp_avg_count*packets.get_payload_sizes().size()) * n_subsets * sizeof(match_type));//just for now
    hipMalloc( (void **) &d_match_count,  (              packets.get_payload_sizes().size()) * n_subsets * sizeof(unsigned int));//just for now
	hipMalloc( (void **) &d_accum_dfa_state_table_lengths, n_subsets * sizeof(unsigned int));
    	
	size_t tmp_dfa_state_table_total_size=0, tmp_curr_dfa_state_table_size=0, tmp_accum_prev_dfa_state_table_size=0;//in bytes
	for (unsigned int i = 0; i < n_subsets; i++) {//Find total size (in bytes) of each data structure
		tmp_dfa_state_table_total_size +=  fa[i]->get_dfa_state_table_size();
	}
	
	//Allocate device memory
	hipMalloc((void **) &d_dfa_state_tables, tmp_dfa_state_table_total_size);
    hipMalloc((void **) &d_input, packets.get_payloads().size() * sizeof(*d_input));
    hipMalloc((void **) &d_pkt_size, packets.get_payload_sizes().size() * sizeof(*d_pkt_size));
	
	for (unsigned int i = 0; i < n_subsets; i++){//Copy to device memory
		hipError_t retval3;
		tmp_curr_dfa_state_table_size =  fa[i]->get_dfa_state_table_size();
				
		if (i==0){
			retval3 = hipMemcpy( d_dfa_state_tables, fa[i]->get_dfa_state_table(), tmp_curr_dfa_state_table_size, hipMemcpyHostToDevice);
		}
		else{
			tmp_accum_prev_dfa_state_table_size +=  fa[i-1]->get_dfa_state_table_size();
			retval3 = hipMemcpy( &d_dfa_state_tables[tmp_accum_prev_dfa_state_table_size/sizeof(state_t)], fa[i]->get_dfa_state_table(), tmp_curr_dfa_state_table_size, hipMemcpyHostToDevice);
		}
		accum_dfa_state_table_lengths[i] = tmp_accum_prev_dfa_state_table_size/sizeof(state_t);
	
		if (retval3 != hipSuccess) cout << "Error while copying dfa state table to device memory" << endl;
	}

    hipError_t retval = hipMemcpy(d_input, &(packets.get_payloads()[0]), packets.get_payloads().size() * sizeof(*d_input), hipMemcpyHostToDevice);
    if (retval != hipSuccess) cout << "Error while copying payload to device memory" << endl;
	
    retval = hipMemcpy(d_pkt_size, &(packets.get_payload_sizes()[0]), packets.get_payload_sizes().size() * sizeof(*d_pkt_size), hipMemcpyHostToDevice);
	if (retval != hipSuccess) cout << "Error while copying packet sizes to device memory" << endl;
	
	hipMemcpy( d_accum_dfa_state_table_lengths, accum_dfa_state_table_lengths,    n_subsets * sizeof(unsigned int), hipMemcpyHostToDevice);
			
	GPUMemInfo();
	
	//Tuning blocksize
	int device;
	hipDeviceProp_t props;
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device); printf("multiProcessorCount = %d\n", props.multiProcessorCount);
	
	int blockSize_, maxActiveBlocks;
	float occupancy;
	int n_packets = packets.get_payload_sizes().size();
	
	blockSize_ = 1;
	hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, udfa_kernel, blockSize_, max_shmem);
	printf("Required computational workload: subsets = %d, packets = %d, subsets*packets = %d\n", n_subsets, n_packets, n_packets * n_subsets);
	//while ( (  blockSize_*maxActiveBlocks*props.multiProcessorCount <= packets.get_payload_sizes().size()*n_subsets ) && (blockSize_ < n_subsets) && (blockSize_ < 1024) ){	//not very correct
	while ( (  maxActiveBlocks*props.multiProcessorCount <= n_packets * (n_subsets/blockSize_ + 1) ) && (blockSize_ < n_subsets) && (blockSize_ < 1024) ){
		occupancy = (maxActiveBlocks * blockSize_ / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize)*100;
		printf("Inter. theoretical GPU launch info: blockSize_ = %d, maxActiveBlocks = %d, maxActiveBlocks*multiProcessorCount = %d, grid.x=%d, grid.y=%d, occupancy: %f\n", blockSize_, maxActiveBlocks, maxActiveBlocks*props.multiProcessorCount, n_packets, n_subsets/blockSize_ + 1, occupancy);
		blockSize_++;
		hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, udfa_kernel, blockSize_, max_shmem);
	}
	*blocksize = blockSize_;
	hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, udfa_kernel, *blocksize, max_shmem);
	occupancy = (maxActiveBlocks * (*blocksize) / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize)*100;	
	printf("Final theoretical GPU launch info: blocksize = %d, maxActiveBlocks = %d, maxActiveBlocks*multiProcessorCount = %d, grid.x=%d, grid.y=%d, occupancy: %f\n", *blocksize, maxActiveBlocks, maxActiveBlocks*props.multiProcessorCount, n_packets, n_subsets/(*blocksize) + 1, occupancy);

#ifdef TEXTURE_MEM_USE
	// bind textures to d_dfa_state_tables
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<state_t>();
    hipBindTexture(0, tex_dfa_state_tables, d_dfa_state_tables, channelDesc, tmp_dfa_state_table_total_size);
    printf("Texture memory usage: %lf MB\n", tmp_dfa_state_table_total_size/1024.0/1024.0);
#endif	
	gettimeofday(&c1, NULL);
	
	// Launch kernel (asynchronously)
	//printf("Size of symbol = %d, Size of unsigned char = %d\n",sizeof(symbol), sizeof(unsigned char));
	printf("U-DFA kernel\n");
	dim3 block(cfg.get_threads_per_block(),1);
	dim3 grid (packets.get_payload_sizes().size(),n_subsets/cfg.get_threads_per_block() + 1);
	
	hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, udfa_kernel, cfg.get_threads_per_block(), max_shmem);
	occupancy = (maxActiveBlocks * cfg.get_threads_per_block() / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize)*100;	
	printf("Manual GPU launch info: blocksize = %d, maxActiveBlocks = %d, maxActiveBlocks*multiProcessorCount = %d, grid.x=%d, grid.y=%d, occupancy: %f\n", cfg.get_threads_per_block(), maxActiveBlocks, maxActiveBlocks*props.multiProcessorCount, grid.x, grid.y, occupancy);
	
	if (blksiz_tuning == 1) {
		block.x = *blocksize;
		grid.x = packets.get_payload_sizes().size();
		grid.y = n_subsets/(*blocksize) + 1;
		printf("Blocksize tuning is being used!\n");
	}
	cout << "GPU launch info: block.x = " << block.x << ", grid.x = " << grid.x << ", grid.y = " << grid.y << endl;

#ifdef TEXTURE_MEM_USE
    printf("Store DFA STATE TABLE in texture memory!\n");
    udfa_kernel_texture<<<grid, block>>>(
                                        (symboln*)d_input,
                                        d_pkt_size, packet_size,
                                        d_match_count, d_match_array, tmp_avg_count,
                                        d_accum_dfa_state_table_lengths, n_subsets);
#else
	printf("Store DFA STATE TABLE in global memory!\n");
    udfa_kernel<<<grid, block>>>(d_dfa_state_tables,
                                (symboln*)d_input,
                                d_pkt_size, packet_size,
                                d_match_count, d_match_array, tmp_avg_count,
                                d_accum_dfa_state_table_lengths, n_subsets);							    
#endif
				
	hipDeviceSynchronize();
	
	gettimeofday(&c2, NULL);

#ifdef TEXTURE_MEM_USE
    // unbind textures from d_nfa_state_tables, d_ptr_state_tables
    hipUnbindTexture(tex_dfa_state_tables);
#endif
	
	seconds  = c2.tv_sec  - c1.tv_sec;
	useconds = c2.tv_usec - c1.tv_usec;
    *t_kernel= ((double)seconds * 1000 + (double)useconds/1000.0);
	printf("host_functions.cu: t_kernel= %lf(ms)\n", *t_kernel);
	
	hipMemcpy( h_match_count, d_match_count,                packets.get_payload_sizes().size()  * n_subsets * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy( h_match_array, d_match_array, (tmp_avg_count*packets.get_payload_sizes().size()) * n_subsets * sizeof(match_type), hipMemcpyDeviceToHost);
	
	// Collect results
	//Temporarily comment the following FOR loop
	unsigned int total_matches=0;
	for (unsigned int i = 0; i < n_subsets; i++) {
#ifdef TEXTURE_MEM_USE
        strcpy (filename,"../bin/data/Report_tex_");
#else
        strcpy (filename,"../bin/data/Report_global_");
#endif
		snprintf(bufftmp, sizeof(bufftmp),"%d",n_subsets);
		strcat (filename,bufftmp);
		strcat (filename,"_");
		snprintf(bufftmp, sizeof(bufftmp),"%d",i+1);
		strcat (filename,bufftmp);
		strcat (filename,".txt");
		fp_report.open (filename); //cout << "Report filename:" << filename << endl;
		fa[i]->mapping_states2rules(&h_match_count[packets.get_payload_sizes().size()*i], &h_match_array[tmp_avg_count*packets.get_payload_sizes().size()*i], 
		                            tmp_avg_count, packets.get_payload_sizes(), packets.get_padded_sizes(), fp_report, rulestartvec, i);
		fp_report.close();
		for (unsigned int j = 0; j < packets.get_payload_sizes().size(); j++)
			total_matches += h_match_count[j + packets.get_payload_sizes().size()*i];		
	}
	printf("Host - Total number of matches %d\n", total_matches);

    gettimeofday(&c3, NULL);
	
	// Free some memory
	hipFree(d_match_count);
	hipFree(d_match_array);
	hipFree(d_accum_dfa_state_table_lengths);
	hipFree(d_dfa_state_tables);
	hipFree(d_input);
    hipFree(d_pkt_size);

	free(h_match_count);
    free(h_match_array);
	free(accum_dfa_state_table_lengths);
		
	gettimeofday(&c4, NULL);
	
	seconds  = c1.tv_sec  - c0.tv_sec;
	useconds = c1.tv_usec - c0.tv_usec;
    *t_alloc = ((double)seconds * 1000 + (double)useconds/1000.0);
	
	seconds  = c2.tv_sec  - c1.tv_sec;
	useconds = c2.tv_usec - c1.tv_usec;
    *t_kernel= ((double)seconds * 1000 + (double)useconds/1000.0);
	
	seconds    = c3.tv_sec  - c2.tv_sec;
	useconds   = c3.tv_usec - c2.tv_usec;
    *t_collect = ((double)seconds * 1000 + (double)useconds/1000.0);

	seconds  = c4.tv_sec  - c3.tv_sec;
	useconds = c4.tv_usec - c3.tv_usec;
    *t_free  = ((double)seconds * 1000 + (double)useconds/1000.0);
	
	return 0;
}
/*--------------------------------------------------------------------------------------------------*/
#ifdef TEXTURE_MEM_USE
__global__ void udfa_kernel_texture(symboln *input,
									unsigned int *pkt_size_vec, unsigned int pkt_size,
									unsigned int *match_count, match_type *match_array, unsigned int match_vec_size,
									unsigned int *accum_dfa_state_table_lengths, unsigned int n_subsets){					
	
	unsigned int dfa_id = threadIdx.x + blockIdx.y * blockDim.x;
	match_type tmp_match;
	
	if (dfa_id >= n_subsets)
		return;
	
	unsigned int shr_match_count = 0;
	
	//cur_pkt_size is the input string length of the packet
	unsigned int cur_pkt_size = pkt_size_vec[blockIdx.x];

	//skip to the right input string
	input += (pkt_size * blockIdx.x/fetch_bytes); 

	unsigned int accum_dfa_state_table_length = accum_dfa_state_table_lengths[dfa_id];
	
	state_t current_state = 0;

	//Payload loop
	for(unsigned int p=0; p<cur_pkt_size; p+=fetch_bytes, input++){
		symboln Input_ = *input;//fetch 4 bytes from the input string
		for (unsigned int byt = 0; byt < fetch_bytes; byt++) {
			unsigned int Input = Input_ & 0xFF;//extract 1 byte
			Input_  = Input_ >> 8;//Input_ right-shifted by 8 bits
				
			//Query the state table on the input symbol for the next state
			current_state = tex1Dfetch(tex_dfa_state_tables, current_state * CSIZE + Input + accum_dfa_state_table_length);
			
			if (current_state < 0) {//Added for matching operation: check if the dst state is an accepting state
				current_state = -current_state;
				//match_offset[match_vec_size*blockIdx.x + shr_match_count + dfa_id*match_vec_size*gridDim.x] = p;
				//match_states[match_vec_size*blockIdx.x + shr_match_count + dfa_id*match_vec_size*gridDim.x] = current_state;
				tmp_match.off  = p + byt;
				tmp_match.stat = current_state;
				match_array[shr_match_count + match_vec_size*(blockIdx.x + dfa_id*gridDim.x)] = tmp_match;
				
				shr_match_count = shr_match_count + 1;			
			}
		}
	}
	match_count[blockIdx.x + dfa_id*gridDim.x] = shr_match_count;
}
#endif